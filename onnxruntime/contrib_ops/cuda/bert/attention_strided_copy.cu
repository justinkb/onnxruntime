#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "contrib_ops/cuda/bert/attention_impl.h"

template <typename T>
__global__ void StridedCopy(const T* in, const int H, longlong4 in_stride,  // coord (b,n,s,h)
                            T* out, longlong4 out_stride                    // coord (b,n,s,h)
) {
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  if (h < H) {
    const int in_offset = b * in_stride.x + n * in_stride.y + s * in_stride.z + h * in_stride.w;
    const int out_offset = b * out_stride.x + n * out_stride.y + s * out_stride.z + h * out_stride.w;
    out[out_offset] = in[in_offset];
  }
}

template <typename T>
__global__ void StridedCopyLarge(const T* in, const int H, longlong4 in_stride,  // coord (b,n,s,h)
                                 T* out, longlong4 out_stride                    // coord (b,n,s,h)
) {
  // Use when (H*)*num_heads > 1024
  int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int h_step = blockDim.x;

  while (h < H) {
    const int in_offset = b * in_stride.x + n * in_stride.y + s * in_stride.z + h * in_stride.w;
    const int out_offset = b * out_stride.x + n * out_stride.y + s * out_stride.z + h * out_stride.w;
    out[out_offset] = in[in_offset];
    h += h_step;
  }
}

template <int NumBytes>
struct ToByteType;

template <>
struct ToByteType<2> {
  using T = uchar2;
};

template <>
struct ToByteType<4> {
  using T = uint;
};

template <>
struct ToByteType<8> {
  using T = uint2;
};

template <>
struct ToByteType<16> {
  using T = uint4;
};

template <>
struct ToByteType<32> {
  using T = ulonglong4;
};

template <int NumBytes>
using ToBytes = typename ToByteType<NumBytes>::T;

template <typename T>
Status LaunchStridedCopy(hipStream_t stream,
                         const T* in, int4 in_shape, longlong4 in_stride,  // coord (b,n,s,h)
                         T* out, longlong4 out_stride,                     // coord (b,n,s,h)
                         int max_threads_per_block) {
  int batch_size = in_shape.x;
  int num_heads = in_shape.y;
  int sequence_length = in_shape.z;
  int head_size = in_shape.w;

  const dim3 grid(sequence_length, batch_size);
  if (0 == (head_size % 4)) { // pack 4 element together
    using Bytes = ToBytes<sizeof(T) * 4>;
    const int H = head_size / 4;
    in_stride.x /= 4;
    in_stride.y /= 4;
    in_stride.z /= 4;
    out_stride.x /= 4;
    out_stride.y /= 4;
    out_stride.z /= 4;
    if (H * num_heads <= max_threads_per_block) {
      const dim3 block(H, num_heads, 1);
      StridedCopy<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), H, in_stride,
                                                     reinterpret_cast<Bytes*>(out), out_stride);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      StridedCopyLarge<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), H, in_stride,
                                                          reinterpret_cast<Bytes*>(out), out_stride);
    }
  } else if (0 == (head_size % 2)) { // pack 2 element together
    using Bytes = ToBytes<sizeof(T) * 2>;
    const int H = head_size / 2;
    in_stride.x /= 2;
    in_stride.y /= 2;
    in_stride.z /= 2;
    out_stride.x /= 2;
    out_stride.y /= 2;
    out_stride.z /= 2;
    if (H * num_heads <= max_threads_per_block) {
      const dim3 block(H, num_heads, 1);
      StridedCopy<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), H, in_stride,
                                                     reinterpret_cast<Bytes*>(out), out_stride);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      StridedCopyLarge<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), H, in_stride,
                                                          reinterpret_cast<Bytes*>(out), out_stride);
    }
  } else {
    using Bytes = ToBytes<sizeof(T)>;
    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      StridedCopy<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), head_size, in_stride,
                                                     reinterpret_cast<Bytes*>(out), out_stride);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      StridedCopyLarge<Bytes><<<grid, block, 0, stream>>>(reinterpret_cast<const Bytes*>(in), head_size, in_stride,
                                                          reinterpret_cast<Bytes*>(out), out_stride);
    }
  }
  return hipGetLastError();
}
